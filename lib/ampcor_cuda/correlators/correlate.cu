#include "hip/hip_runtime.h"
// -*- C++ -*-
//
// michael a.g. aïvázis <michael.aivazis@para-sim.com>
// parasim
// (c) 1998-2020 all rights reserved


// configuration
#include <portinfo>
// STL
#include <complex>
// pyre
#include <pyre/journal.h>
// cuda
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
// pull the declarations
#include "kernels.h"


// we use floats
using value_t = float;


// the correlation kernel
template <std::size_t T, typename valueT = value_t>
__global__
void
_correlate(const valueT * refArena, const valueT * refStats,
           const valueT * secArena, const valueT * secStats,
           std::size_t refRows, std::size_t refCols,
           std::size_t secRows, std::size_t secCols,
           std::size_t corRows, std::size_t corCols,
           std::size_t row, std::size_t col,
           valueT * correlation);


// implementation
void
ampcor::cuda::kernels::
correlate(const value_t * refArena, const value_t * refStats,
          const value_t * secArena, const value_t * secStats,
          std::size_t pairs,
          std::size_t refRows, std::size_t refCols,
          std::size_t secRows, std::size_t secCols,
          std::size_t corRows, std::size_t corCols,
          value_t * dCorrelation)
{
    // make a channel
    pyre::journal::debug_t channel("ampcor.cuda");

    // figure out the job layout and launch the calculation on the device
    // each thread block takes care of one tile pair, so we need as many blocks as there are pairs
    auto B = pairs;
    // the computation of the correlation matrix is a reduction that starts with each thread
    // handling a pair of columns, one from the reference tile and one from a particular
    // placement of the chip within the search window; this means that the number of threads
    // per block is determined by the number of columns in the reference tile
    auto T = refCols;
    // each thread stores in shared memory the partial sum for the numerator term and the
    // partial sum for the secondary tile variance; so we need two {value_t}'s worth of shared
    // memory for each thread, but no less than 64
    auto S = 2 * std::max(T, 64ul) * sizeof(value_t);

    // show me
    channel
        << pyre::journal::at(__HERE__)
        << "launching " << B << " blocks of " << T << " threads each, with "
        << S << " bytes of shared memory per block, for each of the "
        << "(" << corRows << "x" << corCols << ")"
        << " possible placements of the search window within the secondary tile;"
        << " a grand total of " << (B*corRows*corCols) << " kernel launches"
        << pyre::journal::endl;

    // for storing error codes
    hipError_t status = hipSuccess;
    // go through all possible row offsets for the sliding window
    for (auto row = 0; row < corRows; ++row) {
        // and all possible column offsets
        for (auto col = 0; col < corCols; ++col) {
            // deduce the correct kernel to launch and deploy
            // N.B.: kernel launch is an implicit barrier, so no need for any extra
            // synchronization
            if (refCols <= 32) {
                // tell me
                channel << "deploying the 32 column kernel";
                // do it
                _correlate<32> <<<B,32,S>>> (refArena, refStats, secArena, secStats,
                                             refRows, refCols, secRows, secCols, corRows, corCols,
                                             row, col, dCorrelation);
            } else if (refCols <= 64) {
                // tell me
                channel << "deploying the 64 column kernel";
                // do it
                _correlate<64> <<<B,64,S>>> (refArena, refStats, secArena, secStats,
                                             refRows, refCols, secRows, secCols, corRows, corCols,
                                             row, col, dCorrelation);
            } else if (refCols <= 128) {
                // tell me
                channel << "deploying the 128 column kernel";
                // do it
                _correlate<128> <<<B,128,S>>> (refArena, refStats, secArena, secStats,
                                               refRows, refCols, secRows, secCols, corRows, corCols,
                                               row, col, dCorrelation);
            } else if (refCols <= 256) {
                // tell me
                channel << "deploying the 256 column kernel";
                // do it
                _correlate<256> <<<B,256,S>>> (refArena, refStats, secArena, secStats,
                                               refRows, refCols, secRows, secCols, corRows, corCols,
                                               row, col, dCorrelation);
            } else if (refCols <= 512) {
                // tell me
                channel << "deploying the 512 column kernel";
                // do it
                _correlate<512> <<<B,512,S>>> (refArena, refStats, secArena, secStats,
                                               refRows, refCols, secRows, secCols, corRows, corCols,
                                               row, col, dCorrelation);
            } else {
                // complain
                throw std::runtime_error("cannot handle reference tiles of this shape");
            }
            // check for errors
            status = hipPeekAtLastError();
            // if something went wrong
            if (status != hipSuccess) {
                // make a channel
                pyre::journal::error_t error("ampcor.cuda");
                // complain
                error
                    << pyre::journal::at(__HERE__)
                    << "after launching the " << row << "x" << col << " correlators: "
                    << hipGetErrorName(status) << " (" << status << ")"
                    << pyre::journal::endl;
                // and bail
                break;
            }
        }
        // if something went wrong in the inner loop
        if (status != hipSuccess) {
            // bail out of the outer loop as well
            break;
        }
    }
    // wait for the device to finish
    status = hipDeviceSynchronize();
    // check
    if (status != hipSuccess) {
        // get the error description
        std::string description = hipGetErrorName(status);
        // make a channel
        pyre::journal::error_t error("ampcor.cuda");
        // complain
        error
            << pyre::journal::at(__HERE__)
            << "while waiting for a kernel to finish: "
            << description << " (" << status << ")"
            << pyre::journal::endl;
        // and bail
        throw std::runtime_error(description);
    }

    // all done
    return;
}


// the correlation kernel
template <std::size_t T, typename valueT>
__global__
void
_correlate(const valueT * refArena, // the reference tiles
           const valueT * refStats, // the hyper-grid of reference tile variances
           const valueT * secArena, // the secondary tiles
           const valueT * secStats, // the hyper-grid of secondary tile averages
           std::size_t refRows, std::size_t refCols,
           std::size_t secRows, std::size_t secCols,
           std::size_t corRows, std::size_t corCols,
           std::size_t row, std::size_t col,
           valueT * correlation)
{

    // build the workload descriptors
    // global
    // std::size_t B = gridDim.x;    // number of blocks
    // std::size_t T = blockDim.x;   // number of threads per block
    // std::size_t W = B*T;          // total number of workers
    // local
    std::size_t b = blockIdx.x;      // my block id
    std::size_t t = threadIdx.x;     // my thread id within my block
    // std::size_t w = b*T + t;      // my worker id


    // N.B.: do not be tempted to terminate early threads that have no assigned workload; their
    // participation is required to make sure that shared memory is properly zeroed out for the
    // nominally out of bounds accesses

    // get access to my shared memory
    extern __shared__ valueT scratch[];
    // get a handle to this thread block group
    cooperative_groups::thread_block cta = cooperative_groups::this_thread_block();

    // initialize the numerator term
    valueT numerator = 0;
    // initialize the secondary variance accumulator
    valueT secVariance = 0;
    // look up the mean secondary amplitude: skip over the cells handled by other block, and
    // then skip over the cells handled by other threads in my block
    auto mean = secStats[b*corRows*corCols + row*corCols + col];

    // my {ref} starting point is column {t} of grid {b}
    auto ref = refArena + b*refRows*refCols + t;
    // my {sec} starting point is column {t} of the slice of grid {b} at (row, col)
    auto sec = secArena + b*secRows*secCols + (row*secCols + col) + t;

    // if my thread id is less than the number of columns in the reference tile, i need to sum
    // up the contributions to the numerator and the secondary tile variance from my column; if
    // not, my contribution is to zero out my slots in shared memory so the reduction doesn't
    // read uninitialized memory
    if (t < refCols) {
        // run down the two matching columns, one from {ref}, one from {sec}
        for (std::size_t idx=0; idx < refCols; ++idx) {
            // fetch the {ref} value
            valueT r = ref[idx*refCols];
            // fetch the {sec} value and subtract the mean secondary amplitude
            valueT t = sec[idx*secCols] - mean;
            // update the numerator
            numerator += r * t;
            // and the secondary variance
            secVariance += t * t;
        }
    }

    // save my partial results; idle threads only do this bit, with {numerator} and
    // {secVariance} still at their initial values
    scratch[2*t] = numerator;
    scratch[2*t + 1] = secVariance;
    // barrier: make sure everybody is done
    cta.sync();

    // now do the reduction in shared memory
    // for progressively smaller block sizes, the bottom half of the threads collect partial sums
    // N.B.: T is a template parameter, known at compile time, so it's easy for the optimizer to
    // eliminate the impossible clauses
    // for 512 threads per block
    if (T >= 512 && t < 256) {
        // my sibling's offset
        auto offset = 2*(t+256);
        // update my partial sum by reading my sibling's value
        numerator += scratch[offset];
        // ditto for the secondary variance
        secVariance += scratch[offset+1];
        // and make them available
        scratch[2*t] = numerator;
        scratch[2*t+1] = secVariance;
    }
    // make sure everybody is done
    cta.sync();

    // for 256 threads per block
    if (T >= 256 && t < 128) {
        // my sibling's offset
        auto offset = 2*(t+128);
        // update my partial sum by reading my sibling's value
        numerator += scratch[offset];
        // ditto for the secondary variance
        secVariance += scratch[offset+1];
        // and make them available
        scratch[2*t] = numerator;
        scratch[2*t+1] = secVariance;
    }
    // make sure everybody is done
    cta.sync();

    // for 128 threads per block
    if (T >= 128 && t < 64) {
        // my sibling's offset
        auto offset = 2*(t+64);
        // update my partial sum by reading my sibling's value
        numerator += scratch[offset];
        // ditto for the secondary variance
        secVariance += scratch[offset+1];
        // and make them available
        scratch[2*t] = numerator;
        scratch[2*t+1] = secVariance;
    }
    // make sure everybody is done
    cta.sync();

    // on recent architectures, there is a faster way to do the reduction once we reach the
    // warp level; the only cost is that we have to make sure there is enough memory for 64
    // threads, i.e. the shared memory size is bound from below by 64*sizeof(valueT)
    if (t < 32) {
        // if we need to
        if (T >= 64) {
            // my sibling's offset
            auto offset = 2*(t+32);
            // pull a neighbor's value
            numerator += scratch[offset];
            secVariance += scratch[offset+1];
        }
        // get a handle to the active thread group
        cooperative_groups::coalesced_group active = cooperative_groups::coalesced_threads();
        // the power-of-2 threads
        for (int offset = 16; offset > 0; offset >>= 1) {
            // reduce using {shuffle}
            numerator += active.shfl_down(numerator, offset);
            secVariance += active.shfl_down(secVariance, offset);
        }
    }

    // finally, the master thread of each block
    if (t == 0) {
        // looks up the sqrt of the reference tile variance
        valueT refVariance = refStats[b];
        // computes the correlation
        auto corr = numerator / refVariance / std::sqrt(secVariance);
        // computes the slot where this result goes
        std::size_t slot = b*corRows*corCols + row*corCols + col;
        // and writes the sum to the result vector
        correlation[slot] = corr;
    }

    // all done
    return;
}


// end of file
